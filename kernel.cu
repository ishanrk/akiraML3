#include "hip/hip_runtime.h"
﻿#pragma once
#include "kernel.cuh"
#include<random>
#include<algorithm>
#include<iostream>

#include <hip/device_functions.h>


void random_init(float* data, int dim1)
{
    std::random_device rd;  // Seed
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine
    std::uniform_real_distribution<float> dis(0.0f, 1.0f);

    // Fill `data` with random values between 0 and 1
    for (int i = 0; i < dim1; ++i) {
        data[i] = dis(gen);
    }
}

__global__ void vectorAddUM(float* c, float* a, float* b, int dim1)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < dim1)
    {
        c[i] = a[i] + b[i];
        printf("c[%d] = %f\n", i, c[i]);
    }
}
// Helper function for using CUDA to add vectors in parallel.
float* addWithCuda(float* c, float* a, float* b, int dim1)
{
    int id = hipGetDevice(&id);
    int THRDSZ= 5;
    int BLOCKSZ = (int)ceil(dim1 / THRDSZ);
    float* tempa; float* tempb; float* tempc;
    hipMalloc(&tempa, dim1 * sizeof(float));

    hipMalloc(&tempb, dim1 * sizeof(float));

    hipMalloc(&tempc, dim1 * sizeof(float));

    hipMemcpy(tempa, a, dim1*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tempb, b, dim1 * sizeof(float), hipMemcpyHostToDevice);


    vectorAddUM <<<BLOCKSZ, THRDSZ >>> (tempc, tempa, tempb, dim1);

    hipMemcpy(c, tempc, dim1 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    return c;

}

__global__ void multiplyVectorsKernel(float* A, float* B, float* result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        result[idx] = A[idx] * B[idx]; // Multiply corresponding components
    }
}
__global__ void sumVectorKernel(float* result, float* sumResult, int N) {
    __shared__ float partialSum[256]; // Shared memory for partial sums

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadID = threadIdx.x;

    if (idx < N) {
        partialSum[threadID] = result[idx]; // Load values into shared memory
    }
    else {
        partialSum[threadID] = 0.0f; // Handle out-of-bound indices
    }

    __syncthreads();

    // Reduction within the block to sum the values
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadID < stride) {
            partialSum[threadID] += partialSum[threadID + stride];
        }
        __syncthreads();
    }

    // The first thread of the block writes the sum to the global result array
    if (threadID == 0) {
        sumResult[blockIdx.x] = partialSum[0];
    }
}


float dotCUDA(float* vec1, float* vec2, int N) {
    float* d_vec1, * d_vec2, * d_result, * d_sumResult;
    float sum = 0.0f;

    // Allocate memory on device
    hipMalloc(&d_vec1, N * sizeof(float));
    hipMalloc(&d_vec2, N * sizeof(float));
    hipMalloc(&d_result, N * sizeof(float));
    int numBlocks = (N + 255) / 256;
    hipMalloc(&d_sumResult, numBlocks * sizeof(float));  // Partial results for block-wise sum

    // Copy data to device
    hipMemcpy(d_vec1, vec1, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, vec2, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to multiply vectors
    multiplyVectorsKernel << <numBlocks, 256 >> > (d_vec1, d_vec2, d_result, N);

    // Launch kernel to sum the result vector
    sumVectorKernel << <numBlocks, 256 >> > (d_result, d_sumResult, N);

    // Copy partial results back to host
    float* partialResults = new float[numBlocks];
    hipMemcpy(partialResults, d_sumResult, numBlocks * sizeof(float), hipMemcpyDeviceToHost);

    // Sum the partial results to get the final sum
    for (int i = 0; i < numBlocks; ++i) {
        sum += partialResults[i];
    }

    // Clean up
    delete[] partialResults;
    hipFree(d_vec1);
    hipFree(d_vec2);
    hipFree(d_result);
    hipFree(d_sumResult);

    return sum;
}

__global__ void matrixVectorMulKernel(float* A, float* x, float* y, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // Only process rows within bounds
    if (row < M) {
        float sum = 0.0f;
        // Compute the dot product of row `row` of A and vector x
        for (int col = 0; col < N; col++) {
            sum += A[row * N + col] * x[col];
        }
        // Store the result in the output vector y
        y[row] = sum;
    }
}
void matrixVectorMul(float* A, float* x, float* y, int M, int N) {
    float* d_A, * d_x, * d_y;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, M * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block sizes
    int blockSize = 256;
    int numBlocks = (M + blockSize - 1) / blockSize;

    // Launch the kernel
    matrixVectorMulKernel << <numBlocks, blockSize >> > (d_A, d_x, d_y, M, N);

    // Copy the result back to host
    hipMemcpy(y, d_y, M * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
}


