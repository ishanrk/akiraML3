#include "hip/hip_runtime.h"
﻿#pragma once
#include "kernel.cuh"
#include<random>
#include<algorithm>
#include<iostream>

#include <hip/device_functions.h>


void random_init(float* data, int dim1, int dim2)
{
    std::random_device rd;  // Seed
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine
    std::uniform_real_distribution<float> dis(0.0f, 1.0f);

    // Fill `data` with random values between 0 and 1
    for (int i = 0; i < dim1*dim2; ++i) {
        data[i] = dis(gen);
    }
}

__global__ void vectorAddUM(float* c, float* a, float* b, int dim1)
{
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (i < dim1)
    {
        c[i] = a[i] + b[i];
        
    }
}
// Helper function for using CUDA to add vectors in parallel.
float* addWithCuda(float* c, float* a, float* b, int dim1)
{
    int id = hipGetDevice(&id);
    int THRDSZ= 5;
    int BLOCKSZ = (int)ceil((double)dim1 / (double)THRDSZ);
    float* tempa; float* tempb; float* tempc;
    hipMalloc(&tempa, dim1 * sizeof(float));

    hipMalloc(&tempb, dim1 * sizeof(float));

    hipMalloc(&tempc, dim1 * sizeof(float));

    hipMemcpy(tempa, a, dim1*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tempb, b, dim1 * sizeof(float), hipMemcpyHostToDevice);


    vectorAddUM <<<BLOCKSZ, THRDSZ >>> (tempc, tempa, tempb, dim1);

    hipMemcpy(c, tempc, dim1 * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    return c;

}

__global__ void multiplyVectorsKernel(float* A, float* B, float* result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        result[idx] = A[idx] * B[idx]; // Multiply corresponding components
    }
}
__global__ void sumVectorKernel(float* result, float* sumResult, int N) {
    __shared__ float partialSum[256]; // Shared memory for partial sums

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int threadID = threadIdx.x;

    if (idx < N) {
        partialSum[threadID] = result[idx]; // Load values into shared memory
    }
    else {
        partialSum[threadID] = 0.0f; // Handle out-of-bound indices
    }

    __syncthreads();

    // Reduction within the block to sum the values
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadID < stride) {
            partialSum[threadID] += partialSum[threadID + stride];
        }
        __syncthreads();
    }

    // The first thread of the block writes the sum to the global result array
    if (threadID == 0) {
        sumResult[blockIdx.x] = partialSum[0];
    }
}


float dotCUDA(float* vec1, float* vec2, int N) {
    float* d_vec1, * d_vec2, * d_result, * d_sumResult;
    float sum = 0.0f;

    // Allocate memory on device
    hipMalloc(&d_vec1, N * sizeof(float));
    hipMalloc(&d_vec2, N * sizeof(float));
    hipMalloc(&d_result, N * sizeof(float));
    int numBlocks = (N + 255) / 256;
    hipMalloc(&d_sumResult, numBlocks * sizeof(float));  // Partial results for block-wise sum

    // Copy data to device
    hipMemcpy(d_vec1, vec1, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, vec2, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to multiply vectors
    multiplyVectorsKernel << <numBlocks, 256 >> > (d_vec1, d_vec2, d_result, N);

    // Launch kernel to sum the result vector
    sumVectorKernel << <numBlocks, 256 >> > (d_result, d_sumResult, N);

    // Copy partial results back to host
    float* partialResults = new float[numBlocks];
    hipMemcpy(partialResults, d_sumResult, numBlocks * sizeof(float), hipMemcpyDeviceToHost);

    // Sum the partial results to get the final sum
    for (int i = 0; i < numBlocks; ++i) {
        sum += partialResults[i];
    }

    // Clean up
    delete[] partialResults;
    hipFree(d_vec1);
    hipFree(d_vec2);
    hipFree(d_result);
    hipFree(d_sumResult);

    return sum;
}

__global__ void matrixVectorMulKernel(float* A, float* x, float* y, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // Only process rows within bounds
    if (row < M) {
        float sum = 0.0f;
        // Compute the dot product of row `row` of A and vector x
        for (int col = 0; col < N; col++) {
            sum += A[row * N + col] * x[col];
        }
        // Store the result in the output vector y
        y[row] = sum;
    }
}
void matrixVectorMul(float* A, float* x, float* y, int M, int N) {
    float* d_A, * d_x, * d_y;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, M * N * sizeof(float));
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, M * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block sizes
    int blockSize = 256;
    int numBlocks = (M + blockSize - 1) / blockSize;

    // Launch the kernel
    matrixVectorMulKernel << <numBlocks, blockSize >> > (d_A, d_x, d_y, M, N);

    // Copy the result back to host
    hipMemcpy(y, d_y, M * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
}


__global__ void sigmoidKernel(float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not go out of bounds
    if (idx < N) {
        output[idx] = 1.0f / (1.0f + expf(-input[idx]));  // Sigmoid function
    }
}
void applySigmoid(float* input, float* output, int N) {
    float* d_input, * d_output;
    size_t size = N * sizeof(float);

    // Allocate device memory
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Copy data from host to device
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Launch the sigmoid kernel
    int blockSize = 256; // number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize; // number of blocks
    sigmoidKernel << <numBlocks, blockSize >> > (d_input, d_output, N);

    // Check for errors
    hipDeviceSynchronize();

    // Copy the result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

void applyReLU(float* input, float* output, int N) {
    float* d_input, * d_output;
    size_t size = N * sizeof(float);

    // Allocate device memory
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Copy data from host to device
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Launch the ReLU kernel
    int blockSize = 256; // number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize; // number of blocks
    reluKernel << <numBlocks, blockSize >> > (d_input, d_output, N);

    // Check for errors
    hipDeviceSynchronize();

    // Copy the result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}


__global__ void reluKernel(float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not go out of bounds
    if (idx < N) {
        output[idx] = fmaxf(0.0f, input[idx]);  // ReLU function
    }
}
__global__ void softmaxKernel(float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not go out of bounds
    if (idx < N) {
        float sumExp = 0.0f;

        // First, compute the sum of the exponentials (this could be optimized if using shared memory)
        for (int i = 0; i < N; i++) {
            sumExp += expf(input[i]);
        }

        // Now compute the softmax for this index
        output[idx] = expf(input[idx]) / sumExp;
    }
}

void applySoftmax(float* input, float* output, int N) {
    float* d_input, * d_output;
    size_t size = N * sizeof(float);

    // Allocate device memory
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Copy data from host to device
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // Launch the softmax kernel
    int blockSize = 256; // number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize; // number of blocks
    softmaxKernel << <numBlocks, blockSize >> > (d_input, d_output, N);

    // Check for errors
    hipDeviceSynchronize();

    // Copy the result back to host
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}
__global__ void sigmoidGradientKernel(const float* x, float* grad, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure the thread is within the bounds of the input vector
    if (idx < N) {
        // Sigmoid of the input element
        float sigmoid_val = 1.0f / (1.0f + expf(-x[idx]));

        // Gradient of sigmoid: sigmoid(x) * (1 - sigmoid(x))
        grad[idx] = sigmoid_val * (1.0f - sigmoid_val);
    }
}

// Function to invoke the CUDA kernel
void sigmoidGradient(const float* x, float* grad, int N) {
    float* d_x, * d_grad;

    // Allocate memory on the device
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_grad, N * sizeof(float));

    // Copy the input data from host to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size for CUDA kernel
    int blockSize = 256;  // Number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize;  // Number of blocks needed

    // Launch the kernel
    sigmoidGradientKernel << <numBlocks, blockSize >> > (d_x, d_grad, N);

    // Copy the result back from device to host
    hipMemcpy(grad, d_grad, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free the allocated memory on the device
    hipFree(d_x);
    hipFree(d_grad);
}

__global__ void softmaxGradientKernel(const float* x, float* grad, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we're within bounds of the input vector
    if (idx < N) {
        // First, compute the softmax values for the input vector
        float sum_exp = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum_exp += expf(x[i]);
        }

        // Softmax values
        float softmax_val = expf(x[idx]) / sum_exp;

        // Calculate the gradient for the softmax (Jacobian elements)
        for (int j = 0; j < N; ++j) {
            if (idx == j) {
                grad[idx * N + j] = softmax_val * (1.0f - softmax_val); // Diagonal (i == j)
            }
            else {
                grad[idx * N + j] = -softmax_val * expf(x[j]) / sum_exp; // Off-diagonal (i != j)
            }
        }
    }
}

void softmaxGradient(const float* x, float* grad, int N) {
    float* d_x, * d_grad;

    // Allocate memory on the device
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_grad, N * N * sizeof(float));  // N x N matrix for Jacobian

    // Copy the input data from host to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size for CUDA kernel
    int blockSize = 256;  // Number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize;  // Number of blocks needed

    // Launch the kernel
    softmaxGradientKernel << <numBlocks, blockSize >> > (d_x, d_grad, N);

    // Copy the result back from device to host
    hipMemcpy(grad, d_grad, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free the allocated memory on the device
    hipFree(d_x);
    hipFree(d_grad);
}

__global__ void reluGradientKernel(const float* x, float* grad, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure the thread is within the bounds of the input vector
    if (idx < N) {
        // Gradient of ReLU: 1 if x > 0, 0 if x <= 0
        grad[idx] = (x[idx] > 0.0f) ? 1.0f : 0.0f;
    }
}

void reluGradient(const float* x, float* grad, int N) {
    float* d_x, * d_grad;

    // Allocate memory on the device
    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_grad, N * sizeof(float));

    // Copy the input data from host to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block size for CUDA kernel
    int blockSize = 256;  // Number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize;  // Number of blocks needed

    // Launch the kernel
    reluGradientKernel << <numBlocks, blockSize >> > (d_x, d_grad, N);

    // Copy the result back from device to host
    hipMemcpy(grad, d_grad, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free the allocated memory on the device
    hipFree(d_x);
    hipFree(d_grad);
}

__global__ void rowMatrixMulKernel(float* row, float* matrix, float* result, int n, int m) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index of the result vector

    // Check if the thread is within the valid range of result vector's size
    if (col < m) {
        float sum = 0.0f;

        // Compute the dot product of row and the col-th column of matrix
        for (int i = 0; i < n; i++) {
            sum += row[i] * matrix[i * m + col]; // matrix[i * m + col] accesses element at row i, column col
        }

        // Store the result in the corresponding position
        result[col] = sum;
    }
}

#include <iostream>
#include <hip/hip_runtime.h>

void rowMatrixMul(float* row, float* matrix, float* result, int n, int m) {
    float* d_row, * d_matrix, * d_result;

    // Allocate memory on the device
    hipMalloc(&d_row, n * sizeof(float));
    hipMalloc(&d_matrix, n * m * sizeof(float));
    hipMalloc(&d_result, m * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_row, row, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, n * m * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;  // Number of threads per block
    int numBlocks = (m + blockSize - 1) / blockSize;  // Ensure we cover all columns
    rowMatrixMulKernel << <numBlocks, blockSize >> > (d_row, d_matrix, d_result, n, m);

    // Copy result back to host
    hipMemcpy(result, d_result, m * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_row);
    hipFree(d_matrix);
    hipFree(d_result);
}

__global__ void transposeKernel(float* input, float* output, int width, int height) {
    // Calculate row and column index in the transposed output matrix
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Perform transpose only if within matrix bounds
    if (x < width && y < height) {
        // Write transposed value from input to output
        output[x * height + y] = input[y * width + x];
    }
}

void transposeMatrix(float* d_input, float* d_output, int width, int height) {
    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
        (height + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    transposeKernel << <gridSize, blockSize >> > (d_input, d_output, width, height);
    hipDeviceSynchronize();
}
void transposeMatrixCPU(float* input, float* output, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            output[j * rows + i] = input[i * cols + j];
        }
    }
}

__global__ void rsmeKernel(float* pred, float* actual, float* output, int N) {
    extern __shared__ float temp[]; // Use shared memory to hold partial sums
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Each thread computes squared difference for its data point, if within bounds
    temp[tid] = (idx < N) ? (pred[idx] - actual[idx]) * (pred[idx] - actual[idx]) : 0.0f;

    __syncthreads(); // Synchronize threads to prepare for reduction

    // Parallel reduction to sum all squared differences within the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            temp[tid] += temp[tid + stride];
        }
        __syncthreads();
    }

    // First thread of each block writes result to output
    if (tid == 0) {
        atomicAdd(output, temp[0]); // Atomic add across blocks
    }
}

// Host function to compute RMSE
float computeRMSE(float* pred, float* actual, int N) {
    float* d_pred, * d_actual, * d_output;
    float output = 0.0f;
    float* h_output = &output;

    hipMalloc((void**)&d_pred, N * sizeof(float));
    hipMalloc((void**)&d_actual, N * sizeof(float));
    hipMalloc((void**)&d_output, sizeof(float));
    hipMemcpy(d_pred, pred, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_actual, actual, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output, h_output, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with one block and threads, each block handling part of the array
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    rsmeKernel << <numBlocks, blockSize, blockSize * sizeof(float) >> > (d_pred, d_actual, d_output, N);

    hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Calculate RMSE

    output = sqrtf(output / N);

    // Free memory
    hipFree(d_pred);
    hipFree(d_actual);
    hipFree(d_output);

    return output;
}

__global__ void rsmeDerivativeKernel(float* pred, float* actual, float* grad, int N, float RMSE) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        grad[idx] = (pred[idx] - actual[idx]) / (N*RMSE);
    }
}

// Host function to compute RMSE derivative
void computeRMSEDerivative(float* pred, float* actual, float* grad, int N, float RMSE) {
    float* d_pred, * d_actual, * d_grad;

    hipMalloc((void**)&d_pred, N * sizeof(float));
    hipMalloc((void**)&d_actual, N * sizeof(float));
    hipMalloc((void**)&d_grad, N * sizeof(float));
    hipMemcpy(d_pred, pred, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_actual, actual, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    rsmeDerivativeKernel << <numBlocks, blockSize >> > (d_pred, d_actual, d_grad, N, RMSE);

    hipMemcpy(grad, d_grad, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_pred);
    hipFree(d_actual);
    hipFree(d_grad);
}
__global__ void elementwiseMultiplyKernel(float* x, float* y, float* result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        result[idx] = x[idx] * y[idx];
    }
}

void elementwiseMultiply(float* x, float* y, float* result, int N) {
    float* d_x, * d_y, * d_result;

    hipMalloc((void**)&d_x, N * sizeof(float));
    hipMalloc((void**)&d_y, N * sizeof(float));
    hipMalloc((void**)&d_result, N * sizeof(float));

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    elementwiseMultiplyKernel << <numBlocks, blockSize >> > (d_x, d_y, d_result, N);

    hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
}