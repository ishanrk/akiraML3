#include "variable.cuh"

variable::variable(int dimension1, int dimension2, bool random, std::vector<variable*>currChildren)
{
    dim1 = dimension1;
    dim2 = dimension2;
    rand = random;
    children = currChildren;
    if (currChildren.empty()) { children.push_back(this); }

    if (rand)
    {
        if (dim2 == 1)
        {
            data = (float*)malloc(dim1 * sizeof(float));
            gradientChild1 = (float*)malloc(dim1 * sizeof(float));
            
            random_init(data, dim1);
            // Set all values in `gradient` to 0
            std::fill(gradientChild1, gradientChild1 + dim1, 0.0f);
        }
    }
}

int variable::setData(float* arr, int dimension1)
{
    if (dimension1 != dim1)
    {
        
    }
    else
    {
        for (int x = 0; x < dimension1;x++)
        {
            this->data[x] = arr[x];
        }
    }
    return 1;
}

variable::~variable()
{
  
}
variable variable::operator+(const variable& other) const {
    // Check if dimensions match
    if (this->dim1 != other.dim1 || this->dim2 != other.dim2) {
        throw std::invalid_argument("Dimensions must match for addition.");
    }

    // Create a new variable to store the result with dim1 and dim2 dimensions
    std::vector<variable*> temp;
    temp.push_back(const_cast<variable*>(this));
    temp.push_back(const_cast<variable*>(&other));
    variable result(this->dim1, this->dim2, false, temp);

    // Allocate memory for `result` data and gradient on the GPU using hipMallocManaged
    result.data = (float*)malloc(dim1 * sizeof(float));
    result.gradientChild1 = (float*)malloc(dim1 * sizeof(float));
    result.gradientChild2 = (float*)malloc(dim1 * sizeof(float));

    // Perform element-wise addition
    
    result.data = addWithCuda(result.data, this->data, other.data, dim1);
    
    // Set `gradient` of the new variable to 0
    std::fill(result.gradientChild1, result.gradientChild1 + dim1, 1.0f);
    std::fill(result.gradientChild2, result.gradientChild2 + dim1, 1.0f);

    // Add both operands as children to the result

    return result;
}

// Display function to print data
void variable::print() const {
    
    std::cout << "[";
    for (int i = 0; i < dim1; i++) {
        std::cout << this->data[i];
        if (i < dim1 - 1) std::cout << ", ";
    }
    std::cout << "]" << std::endl;
}

void variable::getChildren()
{
    for (auto i : this->children)
    {
        i->print();
    }
}

variable variable::dot(const variable& other) const
{
    if (this->dim1 != other.dim1 || this->dim2 != other.dim2) {
        throw std::invalid_argument("Dimensions must match for addition.");
    }
    std::vector<variable*> temp;
    temp.push_back(const_cast<variable*>(this));
    temp.push_back(const_cast<variable*>(&other));
    variable result(1, 1, false, temp);

    result.data = (float*)malloc(sizeof(float));
    result.gradientChild1 = (float*)malloc(dim1 * sizeof(float));
    result.gradientChild2 = (float*)malloc(dim1 * sizeof(float));
    hipMemcpy(result.gradientChild1, other.data, dim1 * sizeof(float), hipMemcpyHostToHost);
    hipMemcpy(result.gradientChild2, this->data, dim1 * sizeof(float), hipMemcpyHostToHost);
    *(result.data) = dotCUDA(this->data, other.data, dim1);

    return result;

}

